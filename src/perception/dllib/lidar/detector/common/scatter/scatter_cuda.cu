#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2022 The Helios-X Authors. All Rights Reserved.
 * Author: Ricky Song
 * Time: 2022-1-24
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

//headers in local files
#include "scatter_cuda.h"

__global__ void scatter_kernel( int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature,
                                const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;
    int x_ind = x_coors[i_pillar];
    int y_ind = y_coors[i_pillar];
    float feature = pfe_output[i_feature*MAX_NUM_PILLARS_ + i_pillar];
    scattered_feature[i_feature*GRID_Y_SIZE*GRID_X_SIZE + y_ind * GRID_X_SIZE + x_ind] = feature;
}

ScatterCuda::ScatterCuda(const int NUM_THREADS, const int MAX_NUM_PILLARS, const int GRID_X_SIZE, const int GRID_Y_SIZE):
NUM_THREADS_(NUM_THREADS),
MAX_NUM_PILLARS_(MAX_NUM_PILLARS),
GRID_X_SIZE_(GRID_X_SIZE),
GRID_Y_SIZE_(GRID_Y_SIZE)
{
}

void ScatterCuda::doScatterCuda(const int pillar_count, int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature)
{
  scatter_kernel<<<pillar_count, NUM_THREADS_>>>(x_coors, y_coors, pfe_output, scattered_feature,
                                                MAX_NUM_PILLARS_, GRID_X_SIZE_, GRID_Y_SIZE_);
}



// For (N * channel) tensors ===>  (channel, BEV_H, BEV_W) tensors
__global__ void scatter_kernelV2( int *coors, float *pfe_output, float *scattered_feature,
                                const int FEATURE_NUM, const int BEV_H, const int BEV_W)
{
    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;
    int bev_ind = coors[i_pillar];
    // if (i_feature ==60 && i_pillar % 100 ==0)
    //     printf("Block %d / %d, Thread %d / %d,  bev_ind %d \n", i_pillar, gridDim.x, i_feature, blockDim.x,bev_ind);

    if(bev_ind >= 0) {
        // // int x_ind = x_coors[i_pillar];
        // // int y_ind = y_coors[i_pillar];
        // // pfe_output : N * 64, get  current feature value ;
        float feature = pfe_output[i_pillar*FEATURE_NUM + i_feature];
        // scattered_feature[i_feature*BEV_H*BEV_W + y_ind * BEV_W + x_ind] = feature;
        scattered_feature[i_feature * BEV_H * BEV_W + bev_ind] = feature;
    }

}


ScatterCudaV2::ScatterCudaV2(const int NUM_THREADS, const int FEATURE_NUM, const int GRID_X_SIZE, const int GRID_Y_SIZE):
        NUM_THREADS_(NUM_THREADS),
        FEATURE_NUM_(FEATURE_NUM),
        GRID_X_SIZE_(GRID_X_SIZE),
        GRID_Y_SIZE_(GRID_Y_SIZE)
{
}

// MAX_PILLARS, dev_coors_,  static_cast<float*>(buffers.getHostBuffer("47")), dev_scattered_feature_)
// NUM_THREADS_ need to be consistent with channels of pfe output , default is 64
void ScatterCudaV2::doScatterCuda(const int pillar_count, int *coors, float *pfe_output, float *scattered_feature)
{
    scatter_kernelV2<<<pillar_count, NUM_THREADS_>>>(coors, pfe_output, scattered_feature,
                                                   FEATURE_NUM_, GRID_X_SIZE_, GRID_Y_SIZE_);
}



__global__ void scatter_kernel_multi_heads(int *x_coors, int *y_coors, float *pfe_output,
                               float *scattered_feature, const int grid_x_size,
                               const int grid_y_size) {
    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;
    int x_ind = x_coors[i_pillar];
    int y_ind = y_coors[i_pillar];
    float feature = pfe_output[i_pillar * 64 + i_feature];
    scattered_feature[i_feature * grid_y_size * grid_x_size +
                      y_ind * grid_x_size + x_ind] = feature;
}

ScatterMultiHeadsCuda::ScatterMultiHeadsCuda(const int num_threads, const int grid_x_size,
                         const int grid_y_size)
        : num_threads_(num_threads),
          grid_x_size_(grid_x_size),
          grid_y_size_(grid_y_size) {}

void ScatterMultiHeadsCuda::DoScatterCuda(const int pillar_count, int *x_coors,
                                int *y_coors, float *pfe_output,
                                float *scattered_feature) {
    scatter_kernel_multi_heads<<<pillar_count, num_threads_>>>(x_coors, y_coors, pfe_output,
                                                   scattered_feature,
                                                   grid_x_size_, grid_y_size_);
}




